#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <time.h>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
static void HandleError(hipError_t err, const char *file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define cudaCheckErrors(msg) \
	do { \
		hipError_t __err = hipGetLastError(); \
		if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
				msg, hipGetErrorString(__err), \
				__FILE__, __LINE__); \
			fprintf(stderr, "*** FAILED - ABORTING\n"); \
			return 1; \
		} \
	} while (0)

//for __syncthreads()
#ifndef __HIPCC__
#define __HIPCC__
#endif // !(__HIPCC__)
#include <hip/device_functions.h>

#define ALIVE 1
#define DEAD 0

#define threads 32


__device__ int diff = 0;

__global__ void halo_rows(int dim, char* grid)
{
	// We want id ∈ [1,dim]
	int id = blockDim.x * blockIdx.x + threadIdx.x + 1;

	if (id <= dim)
	{
		//Copy first real row to bottom ghost row
		grid[(dim + 2) * (dim + 1) + id] = grid[(dim + 2) + id];
		//Copy last real row to top ghost row
		grid[id] = grid[(dim + 2) * dim + id];
	}
}
__global__ void halo_columns(int dim, char* grid)
{
	// We want id ∈ [0,dim+1]
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id <= dim + 1)
	{
		//Copy first real column to right most ghost column
		grid[id * (dim + 2) + dim + 1] = grid[id * (dim + 2) + 1];
		//Copy last real column to left most ghost column
		grid[id * (dim + 2)] = grid[id * (dim + 2) + dim];
	}
}

__global__ void GOL(int dim, char* grid, char* gridAfter)
{
	int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
	int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;

	int i = threadIdx.y;
	int j = threadIdx.x;


	// Declare the shared memory 
	__shared__ char gridBefore[threads][threads];

	
	if (ix <= dim + 1 && iy <= dim + 1)
		gridBefore[i][j] = grid[iy * (dim + 2) + ix];

	//Sync all threads in block
	__syncthreads();
	int sum = 0;
	if (iy <= dim && ix <= dim) {
		if (i != 0 && i != blockDim.y - 1 && j != 0 && j != blockDim.x - 1) {

			// Get the sum of neighbors 
			sum += gridBefore[i + 1][j]; //north
			sum += gridBefore[i - 1][j]; //south
			sum += gridBefore[i][j + 1];//east
			sum += gridBefore[i][j - 1]; //west
			sum += gridBefore[i + 1][j + 1];//northeast
			sum += gridBefore[i - 1][j - 1]; //northwest
			sum += gridBefore[i - 1][j + 1];//southeast
			sum += gridBefore[i + 1][j - 1];//southwest

			if (gridBefore[i][j] == ALIVE && sum < 2)
			{
				gridAfter[iy * (dim + 2) + ix] = DEAD;
				diff++;
			}
			else if (gridBefore[i][j] == ALIVE && (sum == 2 || sum == 3))
			{
				gridAfter[iy * (dim + 2) + ix] = ALIVE;
			}
			else if (gridBefore[i][j] == ALIVE && sum > 3)
			{
				gridAfter[iy * (dim + 2) + ix] = DEAD;
				diff++;
			}

			else if (gridBefore[i][j] == DEAD && sum == 3)
			{
				gridAfter[iy * (dim + 2) + ix] = ALIVE;
				diff++;
			}

			else {
				gridAfter[iy * (dim + 2) + ix] = gridBefore[i][j];
			}
		
		}
	}
}




int main(int argc, char* argv[])
{
	int i, j, iter, host_diff;
	char* h_grid;//Grid on host
	char* d_grid; //Grid on device
	char* d_gridAfter; //Second grid used on device only
	char* d_tmpGrid; //temporary grid pointer for swap

	int dim = 840; //Linear dimension of our grid - not counting ghost cells
	int generations = 1000; //Number of game steps

	// Allocate host Grid 
	h_grid = (char*)malloc(sizeof(char)*(dim + 2) * (dim + 2));
	
	// Allocate device grids
	hipMalloc(&d_grid, sizeof(char)*(dim + 2) * (dim + 2));
	cudaCheckErrors("malloc failed");

	hipMalloc(&d_gridAfter, sizeof(char)*(dim + 2) * (dim + 2));
	cudaCheckErrors("malloc failed");
	int psb;
	// initialize with possibilities
	for (i = 1; i <= dim; i++) {
		for (j = 1; j <= dim; j++) {
			psb = rand() % 100 + 1;
			if (psb <= 40)
				h_grid[i * (dim + 2) + j] = '1';
			else
				h_grid[i * (dim + 2) + j] = '0';
		}
	}

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(GOL), hipFuncCachePreferShared);

	// Copy over initial game grid (Dim-1 threads)
	hipMemcpy(d_grid, h_grid, sizeof(char)*(dim + 2) * (dim + 2), hipMemcpyHostToDevice);
	dim3 blockSize(threads, threads, 1);
	int linGrid_x = (int)ceil(dim / (float)(threads - 2));
	int linGrid_y = (int)ceil(dim / (float)(threads - 2));
	dim3 gridSize(linGrid_x, linGrid_y, 1);

	dim3 cpyBlockSize(threads, 1, 1);
	dim3 cpyGridRowsGridSize((int)ceil(dim / (float)cpyBlockSize.x), 1, 1);
	dim3 cpyGridColsGridSize((int)ceil((dim + 2) / (float)cpyBlockSize.x), 1, 1);

	int counter = 0;
	float elapsed = 0;
	hipEvent_t start, stop;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	HANDLE_ERROR(hipEventRecord(start, 0));
	// Main game loop
	for (iter = 0; iter < generations; iter++) {
		host_diff = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(&diff), &host_diff, sizeof(int), 0, hipMemcpyHostToDevice);
		halo_rows << <cpyGridRowsGridSize, cpyBlockSize >> > (dim, d_grid);
		halo_columns << <cpyGridColsGridSize, cpyBlockSize >> > (dim, d_grid);
		GOL << <gridSize, blockSize >> > (dim, d_grid, d_gridAfter);
		hipMemcpyFromSymbol(&host_diff, HIP_SYMBOL(&diff), sizeof(int), 0, hipMemcpyDeviceToHost);
		counter++;
		if (counter == 10)
		{
			if (host_diff == 0) {
				printf("No change or empty in %d\n", iter+1);
				break;
			}
			counter = 0;
		}
	
	// Swap
	d_tmpGrid = d_grid;
	d_grid = d_gridAfter;
	d_gridAfter = d_tmpGrid;
}
	
		

	// Copy back results and sum
	hipMemcpy(h_grid, d_grid, sizeof(char)*(dim + 2) * (dim + 2), hipMemcpyDeviceToHost);
	
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	printf("The elapsed time in gpu was %f\t sec\n", elapsed/1000);
	// Sum up alive cells and print results
	int total = 0;
	for (i = 1; i <= dim; i++) {
		for (j = 1; j <= dim; j++) {
			total += h_grid[i * (dim + 2) + j];
		}
	}
	printf("Total Alive: %d\n", total);

	hipFree(d_grid);
	hipFree(d_gridAfter);
	free(h_grid);
	

	return 0;
}

